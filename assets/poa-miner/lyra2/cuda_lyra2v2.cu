#include "hip/hip_runtime.h"
/*
* Lyra2 (v2) CUDA Implementation
*
* Based on tpruvot/djm34/VTC sources and incredible 2x boost by Nanashi Meiyo-Meijin (May 2016)
*/

#include <cstdio>
#include <memory.h>
#include "cuda_helper.h"
#include "cuda_lyra2v2_sm3.cuh"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#define __CUDA_ARCH__ 500
#endif

#define TPB5x 128
#define TPB5x2 32

#if __CUDA_ARCH__ >= 500

#include "cuda_lyra2_vectors.h"

#define Nrow 4
#define Ncol 4
#define memshift 3

__device__ uint2x4 *DMatrix;

static __device__ __forceinline__ uint2 LD4S(uint2 *shared_mem, const int index)
{
	return shared_mem[(index * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x];
}

static __device__ __forceinline__ void ST4S(uint2 *shared_mem, const int index, const uint2 data)
{
	shared_mem[(index * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x] = data;
}

static __device__ __forceinline__ uint2 shuffle2(uint2 a, uint32_t b, uint32_t c)
{
	return make_uint2(SHFL(a.x, b, c), SHFL(a.y, b, c));
}

static __device__ __forceinline__
void Gfunc_v5(uint2 &a, uint2 &b, uint2 &c, uint2 &d)
{
	a += b; d = eorswap32(a, d);
	c += d; b ^= c; b = ROR24(b);
	a += b; d ^= a; d = ROR16(d);
	c += d; b ^= c; b = ROR2(b, 63);
}

static __device__ __forceinline__
void round_lyra_v5(uint2x4 s[4])
{
	Gfunc_v5(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc_v5(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc_v5(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc_v5(s[0].w, s[1].w, s[2].w, s[3].w);

	Gfunc_v5(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc_v5(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc_v5(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc_v5(s[0].w, s[1].x, s[2].y, s[3].z);
}

static __device__ __forceinline__
void round_lyra_v5(uint2 s[4])
{
	Gfunc_v5(s[0], s[1], s[2], s[3]);
	s[1] = shuffle2(s[1], threadIdx.x + 1, 4);
	s[2] = shuffle2(s[2], threadIdx.x + 2, 4);
	s[3] = shuffle2(s[3], threadIdx.x + 3, 4);
	Gfunc_v5(s[0], s[1], s[2], s[3]);
	s[1] = shuffle2(s[1], threadIdx.x + 3, 4);
	s[2] = shuffle2(s[2], threadIdx.x + 2, 4);
	s[3] = shuffle2(s[3], threadIdx.x + 1, 4);
}

static __device__ __forceinline__
void reduceDuplexRowSetup2(uint2 *shared_mem, uint2 state[4])
{
	uint2 state1[Ncol][3], state0[Ncol][3], state2[3];
	int i, j;

#pragma unroll
	for(int i = 0; i < Ncol; i++)
	{
#pragma unroll
		for(j = 0; j < 3; j++)
			state0[Ncol - i - 1][j] = state[j];
		round_lyra_v5(state);
	}

	//#pragma unroll 4
	for(i = 0; i < Ncol; i++)
	{
#pragma unroll
		for(j = 0; j < 3; j++)
			state[j] ^= state0[i][j];

		round_lyra_v5(state);

#pragma unroll
		for(j = 0; j < 3; j++)
			state1[Ncol - i - 1][j] = state0[i][j];

#pragma unroll
		for(j = 0; j < 3; j++)
			state1[Ncol - i - 1][j] ^= state[j];
	}

	for(i = 0; i < Ncol; i++)
	{
		const uint32_t s0 = memshift * Ncol * 0 + i * memshift;
		const uint32_t s2 = memshift * Ncol * 2 + memshift * (Ncol - 1) - i*memshift;

#pragma unroll
		for(j = 0; j < 3; j++)
			state[j] ^= state1[i][j] + state0[i][j];

		round_lyra_v5(state);

#pragma unroll
		for(j = 0; j < 3; j++)
			state2[j] = state1[i][j];

#pragma unroll
		for(j = 0; j < 3; j++)
			state2[j] ^= state[j];

#pragma unroll
		for(j = 0; j < 3; j++)
			ST4S(shared_mem, s2 + j, state2[j]);

		uint2 Data0 = shuffle2(state[0], threadIdx.x - 1, 4);
		uint2 Data1 = shuffle2(state[1], threadIdx.x - 1, 4);
		uint2 Data2 = shuffle2(state[2], threadIdx.x - 1, 4);

		if(threadIdx.x == 0)
		{
			state0[i][0] ^= Data2;
			state0[i][1] ^= Data0;
			state0[i][2] ^= Data1;
		}
		else
		{
			state0[i][0] ^= Data0;
			state0[i][1] ^= Data1;
			state0[i][2] ^= Data2;
		}

#pragma unroll
		for(j = 0; j < 3; j++)
			ST4S(shared_mem, s0 + j, state0[i][j]);

#pragma unroll
		for(j = 0; j < 3; j++)
			state0[i][j] = state2[j];

	}

	for(i = 0; i < Ncol; i++)
	{
		const uint32_t s1 = memshift * Ncol * 1 + i*memshift;
		const uint32_t s3 = memshift * Ncol * 3 + memshift * (Ncol - 1) - i*memshift;

#pragma unroll
		for(j = 0; j < 3; j++)
			state[j] ^= state1[i][j] + state0[Ncol - i - 1][j];

		round_lyra_v5(state);

#pragma unroll
		for(j = 0; j < 3; j++)
			state0[Ncol - i - 1][j] ^= state[j];

#pragma unroll
		for(j = 0; j < 3; j++)
			ST4S(shared_mem, s3 + j, state0[Ncol - i - 1][j]);

		uint2 Data0 = shuffle2(state[0], threadIdx.x - 1, 4);
		uint2 Data1 = shuffle2(state[1], threadIdx.x - 1, 4);
		uint2 Data2 = shuffle2(state[2], threadIdx.x - 1, 4);

		if(threadIdx.x == 0)
		{
			state1[i][0] ^= Data2;
			state1[i][1] ^= Data0;
			state1[i][2] ^= Data1;
		}
		else
		{
			state1[i][0] ^= Data0;
			state1[i][1] ^= Data1;
			state1[i][2] ^= Data2;
		}

#pragma unroll
		for(j = 0; j < 3; j++)
			ST4S(shared_mem, s1 + j, state1[i][j]);
	}
	__syncthreads();
}

static __device__
void reduceDuplexRowt2(uint2 *shared_mem, const int rowIn, const int rowInOut, const int rowOut, uint2 state[4])
{
	uint2 state1[3], state2[3];
	const uint32_t ps1 = memshift * Ncol * rowIn;
	const uint32_t ps2 = memshift * Ncol * rowInOut;
	const uint32_t ps3 = memshift * Ncol * rowOut;

	for(int i = 0; i < Ncol; i++)
	{
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 + i*memshift;
		const uint32_t s3 = ps3 + i*memshift;

#pragma unroll
		for(int j = 0; j < 3; j++)
			state1[j] = LD4S(shared_mem, s1 + j);

#pragma unroll
		for(int j = 0; j < 3; j++)
			state2[j] = LD4S(shared_mem, s2 + j);

#pragma unroll
		for(int j = 0; j < 3; j++)
			state[j] ^= state1[j] + state2[j];

		round_lyra_v5(state);

		uint2 Data0 = shuffle2(state[0], threadIdx.x - 1, 4);
		uint2 Data1 = shuffle2(state[1], threadIdx.x - 1, 4);
		uint2 Data2 = shuffle2(state[2], threadIdx.x - 1, 4);

		if(threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

#pragma unroll
		for(int j = 0; j < 3; j++)
			ST4S(shared_mem, s2 + j, state2[j]);
		__syncthreads();

#pragma unroll
		for(int j = 0; j < 3; j++)
			ST4S(shared_mem, s3 + j, LD4S(shared_mem, s3 + j) ^ state[j]);
		__syncthreads();
	}
}

static __device__
void reduceDuplexRowt2x4(uint2 *shared_mem, const int rowInOut, uint2 state[4])
{
	const int rowIn = 2;
	const int rowOut = 3;

	int i, j;
	uint2 last[3];
	const uint32_t ps1 = memshift * Ncol * rowIn;
	const uint32_t ps2 = memshift * Ncol * rowInOut;

#pragma unroll
	for(int j = 0; j < 3; j++)
		last[j] = LD4S(shared_mem, ps2 + j);

#pragma unroll
	for(int j = 0; j < 3; j++)
		state[j] ^= LD4S(shared_mem, ps1 + j) + last[j];

	round_lyra_v5(state);

	uint2 Data0 = shuffle2(state[0], threadIdx.x - 1, 4);
	uint2 Data1 = shuffle2(state[1], threadIdx.x - 1, 4);
	uint2 Data2 = shuffle2(state[2], threadIdx.x - 1, 4);

	if(threadIdx.x == 0)
	{
		last[0] ^= Data2;
		last[1] ^= Data0;
		last[2] ^= Data1;
	}
	else
	{
		last[0] ^= Data0;
		last[1] ^= Data1;
		last[2] ^= Data2;
	}

	if(rowInOut == rowOut)
	{
#pragma unroll
		for(j = 0; j < 3; j++)
			last[j] ^= state[j];
	}

	for(i = 1; i < Ncol; i++)
	{
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 + i*memshift;

#pragma unroll
		for(j = 0; j < 3; j++)
			state[j] ^= LD4S(shared_mem, s1 + j) + LD4S(shared_mem, s2 + j);

		round_lyra_v5(state);
	}

#pragma unroll
	for(int j = 0; j < 3; j++)
		state[j] ^= last[j];
}

__global__
__launch_bounds__(TPB5x, 1)
void lyra2v2_gpu_hash_32_1(uint32_t threads, uint2 *inputHash)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	const uint2x4 blake2b_IV[2] = {
		0xf3bcc908UL, 0x6a09e667UL, 0x84caa73bUL, 0xbb67ae85UL,
		0xfe94f82bUL, 0x3c6ef372UL, 0x5f1d36f1UL, 0xa54ff53aUL,
		0xade682d1UL, 0x510e527fUL, 0x2b3e6c1fUL, 0x9b05688cUL,
		0xfb41bd6bUL, 0x1f83d9abUL, 0x137e2179UL, 0x5be0cd19UL
	};

	const uint2x4 Mask[2] = {
		0x00000020UL, 0x00000000UL, 0x00000020UL, 0x00000000UL,
		0x00000020UL, 0x00000000UL, 0x00000001UL, 0x00000000UL,
		0x00000004UL, 0x00000000UL, 0x00000004UL, 0x00000000UL,
		0x00000080UL, 0x00000000UL, 0x00000000UL, 0x01000000UL
	};

	uint2x4 state[4];

	if(thread < threads)
	{
		state[0].x = state[1].x = __ldg(&inputHash[thread + threads * 0]);
		state[0].y = state[1].y = __ldg(&inputHash[thread + threads * 1]);
		state[0].z = state[1].z = __ldg(&inputHash[thread + threads * 2]);
		state[0].w = state[1].w = __ldg(&inputHash[thread + threads * 3]);
		state[2] = blake2b_IV[0];
		state[3] = blake2b_IV[1];

		for(int i = 0; i<12; i++)
			round_lyra_v5(state);

		state[0] ^= Mask[0];
		state[1] ^= Mask[1];

		for(int i = 0; i<12; i++)
			round_lyra_v5(state);

		DMatrix[blockDim.x * gridDim.x * 0 + thread] = state[0];
		DMatrix[blockDim.x * gridDim.x * 1 + thread] = state[1];
		DMatrix[blockDim.x * gridDim.x * 2 + thread] = state[2];
		DMatrix[blockDim.x * gridDim.x * 3 + thread] = state[3];
	}
}

__global__
__launch_bounds__(TPB5x2, 1)
void lyra2v2_gpu_hash_32_2(uint32_t threads)
{
	const uint32_t thread = blockDim.y * blockIdx.x + threadIdx.y;
	extern __shared__ uint2 shared_mem[];
	if(thread < threads)
	{
		uint2 state[4];
		state[0] = ((uint2*)DMatrix)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[1] = ((uint2*)DMatrix)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[2] = ((uint2*)DMatrix)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[3] = ((uint2*)DMatrix)[(3 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];

		reduceDuplexRowSetup2(shared_mem, state);

		uint32_t rowa;
		int prev = 3;

		for(int i = 0; i < 3; i++)
		{
			rowa = SHFL(state[0].x, 0, 4) & 3;
			reduceDuplexRowt2(shared_mem, prev, rowa, i, state);
			prev = i;
		}

		rowa = SHFL(state[0].x, 0, 4) & 3;
		reduceDuplexRowt2x4(shared_mem, rowa, state);

		((uint2*)DMatrix)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[0];
		((uint2*)DMatrix)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[1];
		((uint2*)DMatrix)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[2];
		((uint2*)DMatrix)[(3 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[3];
	}
}

__global__
__launch_bounds__(TPB5x, 1)
void lyra2v2_gpu_hash_32_3(uint32_t threads, uint2 *outputHash)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	uint2x4 state[4];

	if(thread < threads)
	{
		state[0] = __ldg4(&DMatrix[blockDim.x * gridDim.x * 0 + thread]);
		state[1] = __ldg4(&DMatrix[blockDim.x * gridDim.x * 1 + thread]);
		state[2] = __ldg4(&DMatrix[blockDim.x * gridDim.x * 2 + thread]);
		state[3] = __ldg4(&DMatrix[blockDim.x * gridDim.x * 3 + thread]);

		for(int i = 0; i < 12; i++)
			round_lyra_v5(state);

		outputHash[thread + threads * 0] = state[0].x;
		outputHash[thread + threads * 1] = state[0].y;
		outputHash[thread + threads * 2] = state[0].z;
		outputHash[thread + threads * 3] = state[0].w;
	}
}

#else
#include "cuda_helper.h"
__global__ void lyra2v2_gpu_hash_32_1(uint32_t threads, uint2 *inputHash)
{}
__global__ void lyra2v2_gpu_hash_32_2(uint32_t threads)
{}
__global__ void lyra2v2_gpu_hash_32_3(uint32_t threads, uint2 *outputHash)
{}
#endif


__host__
void lyra2v2_cpu_init(int thr_id, uint64_t *d_matrix)
{
	// just assign the device pointer allocated in main loop
#if CUDART_VERSION >= 9000
	CUDA_SAFE_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(lyra2v2_gpu_hash_32_2), hipFuncAttributePreferredSharedMemoryCarveout, 100)); // make Titan V faster
#endif
	CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(DMatrix), &d_matrix, sizeof(uint64_t*), 0, hipMemcpyHostToDevice, gpustream[thr_id]));
	if(opt_debug)
		CUDA_SAFE_CALL(hipDeviceSynchronize());
}

__host__
void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
{
	if(cuda_arch[thr_id] >= 500)
	{

		const uint32_t tpb = TPB5x;

		dim3 grid2((threads + tpb - 1) / tpb);
		dim3 block2(tpb);

		dim3 grid4((threads * 4 + TPB5x2 - 1) / TPB5x2);
		dim3 block4(4, TPB5x2 / 4);

		lyra2v2_gpu_hash_32_1 << < grid2, block2, 0, gpustream[thr_id] >> > (threads, (uint2*)g_hash);
		if(opt_debug)
			CUDA_SAFE_CALL(hipDeviceSynchronize());
		lyra2v2_gpu_hash_32_2 << < grid4, block4, 384 * TPB5x2, gpustream[thr_id] >> > (threads);
		if(opt_debug)
			CUDA_SAFE_CALL(hipDeviceSynchronize());
		lyra2v2_gpu_hash_32_3 << < grid2, block2, 0, gpustream[thr_id] >> > (threads, (uint2*)g_hash);
		if(opt_debug)
			CUDA_SAFE_CALL(hipDeviceSynchronize());

	}
	else
	{

		uint32_t tpb = 16;
		if(cuda_arch[thr_id] >= 350) tpb = TPB35;
		else if(cuda_arch[thr_id] >= 300) tpb = TPB30;
		else if(cuda_arch[thr_id] >= 200) tpb = TPB20;

		dim3 grid((threads + tpb - 1) / tpb);
		dim3 block(tpb);
		lyra2v2_gpu_hash_32_v3 << < grid, block, 0, gpustream[thr_id] >> > (threads, startNounce, (uint2*)g_hash);
		if(opt_debug)
			CUDA_SAFE_CALL(hipDeviceSynchronize());

	}
	CUDA_SAFE_CALL(hipGetLastError());
}